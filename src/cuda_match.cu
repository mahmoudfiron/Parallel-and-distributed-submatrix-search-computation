#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <atomic>
extern "C" {
  #include "types.h"
  #include "cuda_match.h"
}


// Warn only once per MPI process if there is no GPU
static std::atomic<bool> warned{false};

// Device kernel: one thread per (i,j) candidate
__global__ void matchKernel(const int* __restrict__ pic, int N,
                            const int* __restrict__ obj, int n,
                            double threshold,
                            int maxI, int maxJ,
                            int* foundFlag, int* outI, int* outJ)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i > maxI || j > maxJ) return;

  // If someone already found, skip
  if (atomicAdd(foundFlag, 0) != 0) return;

  double sum = 0.0;
  for (int r = 0; r < n && sum < threshold; ++r) {
    int baseP = (i + r) * N + j;
    int baseO = r * n;
    for (int c = 0; c < n && sum < threshold; ++c) {
      int pv = pic[baseP + c];
      int ov = obj[baseO + c];
      // per spec pv in [1..100], divide-by-zero not expected
      sum += fabs((double)(pv - ov) / (double)pv);
    }
  }
  if (sum < threshold) {
    if (atomicCAS(foundFlag, 0, 1) == 0) { *outI = i; *outJ = j; }
  }
}

int cuda_find_match_for_picture(const Picture* P,
                                const ObjectT* objects, int M,
                                double threshold,
                                MatchResult* out)
{
  // 0) No GPU? Tell caller to fall back to CPU path.
int devCount = 0;
  hipError_t err = hipGetDeviceCount(&devCount);
  if (err != hipSuccess || devCount == 0) {
    if (!warned.exchange(true, std::memory_order_relaxed)) {
      std::fprintf(stderr, "[CUDA] No CUDA device: falling back to CPU (OpenMP).\n");
    }
    return 0;
  }

  const int N = P->N;
  const size_t picBytes = (size_t)N * (size_t)N * sizeof(int);

  // 1) Allocate/copy the picture once
  int *d_pic = nullptr;
  if (hipMalloc(&d_pic, picBytes) != hipSuccess) return 0;
  if (hipMemcpy(d_pic, P->a, picBytes, hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(d_pic); return 0;
  }

  // 2) Device-side results
  int *d_found=nullptr, *d_outI=nullptr, *d_outJ=nullptr;
  hipMalloc(&d_found, sizeof(int));
  hipMalloc(&d_outI, sizeof(int));
  hipMalloc(&d_outJ, sizeof(int));

  // 3) Create two streams: SCopy (prefetch), SComp (compute)
  hipStream_t sCopy = nullptr, sComp = nullptr;
  hipStreamCreate(&sCopy);
  hipStreamCreate(&sComp);

  // Helper: find next valid object index (n <= N)
  auto next_valid = [&](int start)->int {
    int k = start;
    while (k < M && objects[k].n > N) ++k;
    return k;
  };

  // 4) Prefetch the first valid object (if any)
  int k0 = next_valid(0);
  if (k0 >= M) {
    // No valid objects at all
    hipStreamDestroy(sCopy); hipStreamDestroy(sComp);
    hipFree(d_pic); hipFree(d_found); hipFree(d_outI); hipFree(d_outJ);
    return 0;
  }

  // Ping-pong buffers for objects
  int *d_objA = nullptr, *d_objB = nullptr;
  size_t bytesA = 0, bytesB = 0;
  bool useA = true; // current buffer toggle

  // Prefetch first object into A
  {
    int n = objects[k0].n;
    bytesA = (size_t)n * (size_t)n * sizeof(int);
    hipMalloc(&d_objA, bytesA);
    hipMemcpyAsync(d_objA, objects[k0].a, bytesA, hipMemcpyHostToDevice, sCopy);
  }

  // 5) Main pipeline over valid objects
  for (int k = k0; k < M; /* advanced below */) {
    const ObjectT* O = &objects[k];
    const int n  = O->n;
    const int maxI = N - n, maxJ = N - n;

    // Make sure the prefetched buffer for this k is ready
    hipStreamSynchronize(sCopy);

    // Choose the ready buffer as "d_obj"
    int* d_obj   = useA ? d_objA : d_objB;

    // Reset device found flag asynchronously on compute stream
    int zero = 0;
    hipMemcpyAsync(d_found, &zero, sizeof(int), hipMemcpyHostToDevice, sComp);

    // Launch kernel on compute stream
    const int tilesX = maxJ + 1;
    const int tilesY = maxI + 1;
    dim3 block(16,16);
    dim3 grid((tilesX + block.x - 1) / block.x,
          (tilesY + block.y - 1) / block.y);


          matchKernel<<<grid, block, 0, sComp>>>(d_pic, N, d_obj, n, threshold,
                                       maxI, maxJ, d_found, d_outI, d_outJ);

// (optional) check launch error immediately
hipError_t kerr = hipGetLastError();
if (kerr != hipSuccess) {
    std::fprintf(stderr, "CUDA kernel launch failed: %s\n", hipGetErrorString(kerr));
}


    // While compute runs, prefetch NEXT valid object into the other buffer
    int kNext = next_valid(k + 1);
    if (kNext < M) {
      const int n2 = objects[kNext].n;
      size_t bytes2 = (size_t)n2 * (size_t)n2 * sizeof(int);
      if (useA) {
        if (d_objB) hipFree(d_objB);
        bytesB = bytes2;
        hipMalloc(&d_objB, bytesB);
        hipMemcpyAsync(d_objB, objects[kNext].a, bytesB, hipMemcpyHostToDevice, sCopy);
      } else {
        if (d_objA) hipFree(d_objA);
        bytesA = bytes2;
        hipMalloc(&d_objA, bytesA);
        hipMemcpyAsync(d_objA, objects[kNext].a, bytesA, hipMemcpyHostToDevice, sCopy);
      }
    }

    // Wait for the kernel to finish
    hipStreamSynchronize(sComp);

    // Check whether we found a match for this object
    int h_found = 0;
    hipMemcpy(&h_found, d_found, sizeof(int), hipMemcpyDeviceToHost);

    if (h_found) {
      int i, j;
      hipMemcpy(&i, d_outI, sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(&j, d_outJ, sizeof(int), hipMemcpyDeviceToHost);

      out->pictureId = P->id;
      out->found     = 1;
      out->objectId  = O->id;
      out->posI      = i;
      out->posJ      = j;

      // Cleanup
      if (d_objA) hipFree(d_objA);
      if (d_objB) hipFree(d_objB);
      hipStreamDestroy(sCopy);
      hipStreamDestroy(sComp);
      hipFree(d_pic); hipFree(d_found); hipFree(d_outI); hipFree(d_outJ);
      return 1;
    }

    // No match: advance to next valid object
    if (kNext >= M) {
      // No more prefetches pending; we’ll drop out after loop
      // free the current buffer we just used
      if (useA) { hipFree(d_objA); d_objA = nullptr; bytesA = 0; }
      else      { hipFree(d_objB); d_objB = nullptr; bytesB = 0; }
      k = kNext; // == M, end loop
    } else {
      // Free the current buffer; swap to the newly prefetched one next iter
      if (useA) { hipFree(d_objA); d_objA = nullptr; bytesA = 0; }
      else      { hipFree(d_objB); d_objB = nullptr; bytesB = 0; }
      useA = !useA; // toggle buffer (the "other" one has the next object)
      k = kNext;
    }
  }

  // If we got here: no object matched on GPU
  if (d_objA) hipFree(d_objA);
  if (d_objB) hipFree(d_objB);
  hipStreamDestroy(sCopy);
  hipStreamDestroy(sComp);
  hipFree(d_pic); hipFree(d_found); hipFree(d_outI); hipFree(d_outJ);
  return 0;
}

